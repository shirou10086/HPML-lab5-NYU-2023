#include <stdio.h>
#include <time.h>
#include <hipDNN.h>

#define H   1024
#define W   1024
#define C   3
#define FH  3
#define FW  3
#define K   64
#define ITER 5

#define checkCUDNN(expression)                                  \
{                                                               \
    hipdnnStatus_t status = (expression);                        \
    if (status != HIPDNN_STATUS_SUCCESS) {                       \
        printf("cuDNN error on line %d: %s\n", __LINE__,        \
               hipdnnGetErrorString(status));                    \
        exit(EXIT_FAILURE);                                     \
    }                                                           \
}

// Function to load image into memory
void loadImageInMem(int h, int w, int c, double *it) {
    for (int ki = 0; ki < c; ++ki) {
        for (int j = 0; j < h; ++j) {
            for (int i = 0; i < w; ++i) {
                it[ki*w*h + j*w + i] = ki * (i+j);
            }
        }
    }
}

int main(int argc, char *argv[]) {
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Allocate host memory
    double *it = (double *)malloc(C * H * W * sizeof(double));
    double *ot = (double *)malloc(K * H * W * sizeof(double));
    double *f = (double *)malloc(K * C * FH * FW * sizeof(double));

    // Allocate device memory
    double *itg, *otg, *gpuf;
    hipMalloc(&itg, C * H * W * sizeof(double));
    hipMalloc(&otg, K * H * W * sizeof(double));
    hipMalloc(&gpuf, K * C * FH * FW * sizeof(double));

    // Initialize host memory
    loadImageInMem(H, W, C, it);

    // Copy input and filter to GPU
    hipMemcpy(itg, it, C * H * W * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpuf, f, K * C * FH * FW * sizeof(double), hipMemcpyHostToDevice);

    // Set input and output tensor descriptors
    hipdnnTensorDescriptor_t input_descriptor, output_descriptor;
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnConvolutionDescriptor_t convolution_descriptor;

    // Create and set tensor descriptors
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));

    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, C, H, W));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, H, W));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE));

    // Find the best convolution algorithm
    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convolution_algorithm));

    // Allocate workspace for cuDNN
    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor, convolution_algorithm, &workspace_bytes));
    void *d_workspace = nullptr;
    hipMalloc(&d_workspace, workspace_bytes);

    // Perform the convolution
    const double alpha = 1.0, beta = 0.0;
    for (int i = 0; i < ITER; ++i) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        checkCUDNN(hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, itg, kernel_descriptor, gpuf, convolution_descriptor, convolution_algorithm, d_workspace, workspace_bytes, &beta, output_descriptor, otg));
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Iteration %d: Convolution execution time: %f milliseconds\n", i, milliseconds);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Copy the output back to host
    hipMemcpy(ot, otg, K * H * W * sizeof(double), hipMemcpyDeviceToHost);

    // Calculate checksum
    double checksum = 0;
    for (int i = 0; i < K * H * W; ++i) {
        checksum += ot[i];
    }
    printf("Checksum: %lf\n", checksum);

    // Cleanup
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipFree(itg);
    hipFree(otg);
    hipFree(gpuf);
    hipFree(d_workspace);
    hipdnnDestroy(cudnn);

    free(it);
    free(ot);
    free(f);

    return 0;
}
