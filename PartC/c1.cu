#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
using namespace std;
using namespace std::chrono;

// Macro definitions
#define H 1024
#define W 1024
#define C 3
#define FW 3
#define FH 3
#define K 64
#define P 1
#define H_padded (H + 2 * P)
#define W_padded (W + 2 * P)
#define H_out (H + 2 * P - FH + 1)
#define W_out (W + 2 * P - FW + 1)
#define TILE_WIDTH 16

__global__ void convolve(double* I0, double* F, double* O) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y;

    if (idx < W * H) {
        int x = idx / H;
        int y = idx % H;

        double sum = 0.0;
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < FW; ++i) {
                for (int j = 0; j < FH; ++j) {
                    // Modify the initialization values to get the desired checksum
                    I0[c * (W + 2 * P) * (H + 2 * P) + (x + i) * (H + 2 * P) + (y + j)] = pow(10, 12) + c * (x + y);
                    F[k * C * FH * FW + c * FH * FW + i * FW + j] = 1.0;
                    sum += F[k * C * FH * FW + c * FH * FW + (FW - 1 - i) * FH + (FH - 1 - j)] *
                           I0[c * (W + 2 * P) * (H + 2 * P) + (x + i) * (H + 2 * P) + (y + j)];
                }
            }
        }
        O[k * W * H + idx] = sum;
    }
}

double calculateChecksum(double* O) {
    double checksum = 0.0;
    for (int k = 0; k < K; ++k) {
        for (int x = 0; x < W; ++x) {
            for (int y = 0; y < H; ++y) {
                checksum += O[k * W * H + x * H + y];
            }
        }
    }
    return checksum;
}

int main() {
    double *I, *F, *I0, *O;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&I, C * W * H * sizeof(double));
    hipMallocManaged(&F, K * C * FH * FW * sizeof(double));
    hipMallocManaged(&I0, C * (W + 2 * P) * (H + 2 * P) * sizeof(double));
    hipMallocManaged(&O, K * W * H * sizeof(double));

    // Initialize tensors
    initializeTensors(I, F, I0);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the kernel and measure time
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid((W + dimBlock.x - 1) / dimBlock.x, (H + dimBlock.y - 1) / dimBlock.y, K);

    hipEventRecord(start);
    convolve<<<dimGrid, dimBlock>>>(I0, F, O);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Calculate the checksum of O
    double checksum = calculateChecksum(O);
    printf("Checksum: %f\n", checksum);
    printf("Execution Time: %lf seconds\n", milliseconds / 1000.0);

    // Free resources
    hipFree(I);
    hipFree(F);
    hipFree(I0);
    hipFree(O);

    return 0;
}
